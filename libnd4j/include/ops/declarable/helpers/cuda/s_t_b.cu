#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver119 on 19.01.18.
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/s_t_b.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchToSpaceCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft) {

    // input [bS, H * blockSize, W * blockSize, iC]
    // output [bS, H * blockSize - cropBottom - cropTop, W * blockSize - cropLeft - cropRight, iC]

    // if (cropTop = cropBottom = cropRight = cropLeft = 0) shapes are the same
    // else:
    // oH -> [cropBottom, iH - cropTop]
    // oW -> [cropLeft,   iH - cropRight]
    // xLen > zLen

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= zLen)
        return;

    shape::index2coords(rank, zShapeInfo + 1, i, zLen, coords);

    const auto zOffset = shape::getOffset(0, zShapeInfo + 1, zShapeInfo + rank + 1, coords, rank);

    coords[1] += cropBottom;
    coords[2] += cropLeft;

    const auto xOffset = shape::getOffset(0, xShapeInfo + 1, xShapeInfo + rank + 1, coords, rank);

    z[zOffset] = x[xOffset];

}

///////////////////////////////////////////////////////////////////
template<typename T>
static void batchToSpaceCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft) {

    batchToSpaceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, cropBottom, cropLeft);
}
BUILD_SINGLE_TEMPLATE(template void batchToSpaceCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
void batchToSpace(nd4j::LaunchContext* context, const NDArray& input, NDArray& output, const uint cropBottom, const uint cropTop, const uint cropLeft, const uint cropRight, const uint blockSize) {

    // [bS*blockSize*blockSize, H/blockSize, W/blockSize, iC] is rearranged/permuted to [bS, oH, oW, iC]
    // oH = H - cropTop  - cropBottom
    // oW = W - cropLeft - cropRight

    NDArray inputRearranged0 = input.reshape(input.ordering(), {blockSize, blockSize, output.sizeAt(0), input.sizeAt(1), input.sizeAt(2), input.sizeAt(3)});
    inputRearranged0.permutei({2, 3,0, 4,1, 5});

    if(input.lengthOf() == output.lengthOf()) {

        output.assign(inputRearranged0);
    }
    else {

        NDArray inputRearranged1 = inputRearranged0.reshape(input.ordering(), {output.sizeAt(0), input.sizeAt(1) * blockSize, input.sizeAt(2) * blockSize, input.sizeAt(3)});

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "batchToSpace");

        NDArray::prepareSpecialUse({&output}, {&inputRearranged1});
        BUILD_SINGLE_SELECTOR(input.dataType(), batchToSpaceCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), inputRearranged1.getSpecialBuffer(), inputRearranged1.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), cropBottom, cropLeft), LIBND4J_TYPES);
        NDArray::registerSpecialUse({&output}, {&inputRearranged1});

        manager.synchronize();
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void spaceToBatchCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight) {

    // input [bS, H * blockSize - padBottom - padTop, W * blockSize - padLeft - padRight, iC]
    // output [bs, H * blockSize, W * blockSize, iC]

    // if (padTop = padBottom = padRight = padLeft = 0) shapes are the same
    // else:
    // iH -> [padBottom, oH - padTop]
    // iW -> [padLeft,   oW - padRight]
    // zLen > xLen

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= zLen)
        return;

    shape::index2coords(rank, zShapeInfo + 1, i, zLen, coords);

    const auto zOffset = shape::getOffset(0, zShapeInfo + 1, zShapeInfo + rank + 1, coords, rank);

    if(coords[1] >= padBottom && coords[1] < zShapeInfo[2] - padTop && coords[2] >= padLeft && coords[2] < zShapeInfo[3] - padRight) {

        coords[1] -= padBottom;
        coords[2] -= padLeft;

        const auto xOffset = shape::getOffset(0, xShapeInfo + 1, xShapeInfo + rank + 1, coords, rank);

        z[zOffset] = x[xOffset];
    }
    else
        z[zOffset] = 0.f;
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void spaceToBatchCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight) {

    spaceToBatchCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, padBottom, padTop, padLeft, padRight);
}
BUILD_SINGLE_TEMPLATE(template void spaceToBatchCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
void spaceToBatch(nd4j::LaunchContext* context, const NDArray& input, NDArray& output, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight, const uint blockSize) {

    // [bS, iH, iW, iC] is rearranged/permuted to [bS*blockSize*blockSize, (iH + padBottom + padTop)/blockSize, (iW + padLeft + padRight)/blockSize, iC]

    NDArray outputRearranged0 = output.reshape(output.ordering(), {blockSize, blockSize, input.sizeAt(0), output.sizeAt(1), output.sizeAt(2), input.sizeAt(3)});
    outputRearranged0.permutei({2, 3,0, 4,1, 5});

    if(input.lengthOf() == output.lengthOf()) {

        outputRearranged0.assign(input);
    }
    else {

        NDArray outputRearranged1 = outputRearranged0.reshape(output.ordering(), {input.sizeAt(0), output.sizeAt(1) * blockSize, output.sizeAt(2) * blockSize, input.sizeAt(3)});

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "spaceToBatch");

        NDArray::prepareSpecialUse({&outputRearranged1}, {&input});
        BUILD_SINGLE_SELECTOR(input.dataType(), spaceToBatchCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), outputRearranged1.specialBuffer(), outputRearranged1.specialShapeInfo(), padBottom, padTop, padLeft, padRight), LIBND4J_TYPES);
        NDArray::registerSpecialUse({&outputRearranged1}, {&input});

        manager.synchronize();

        if(output.getSpecialBuffer() != outputRearranged1.getSpecialBuffer())
            outputRearranged0.assign(outputRearranged1);
    }
}


/*
    template <int N, bool B2S>
    struct SpaceToBatchHelper {
        template <typename T>
        static void run(T *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, T *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
            for (int batch_pos = 0; batch_pos < batch_shape[0]; ++batch_pos) {
                const int space_pos = batch_pos * block_shape[0] + block_offsets[0] - pad_start[0];
                if (space_pos >= 0 && space_pos < space_shape[0]) {
                    SpaceToBatchHelper<N - 1, B2S>::run(ptrSpace + space_pos * space_strides[0], space_shape + 1, space_strides + 1, block_shape + 1, pad_start + 1, block_offsets + 1, ptrBatch, batch_shape + 1, batch_strides + 1);
                } else {
                    if (!B2S)
                        for (int i = 0; i < batch_strides[0]; i++)
                            ptrBatch[i] = (T) 0.f;
                }

                ptrBatch += batch_strides[0];
            }
        }
    };

    template <bool B2S>
    struct SpaceToBatchHelper<0, B2S> {
        template <typename T>
        static void run(T *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, T *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
            int str = batch_strides[-1];
            for (int i = 0; i < str; i++)
                if (B2S)
                    ptrSpace[i] = ptrBatch[i];
                else
                    ptrBatch[i] = ptrSpace[i];
        }
    };

    template <typename T, int NUM_BLOCK_DIMS, bool B2S>
    void _execute(nd4j::LaunchContext * context, void *vptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, void *vptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
        auto ptrSpace = reinterpret_cast<T *>(vptrSpace);
        auto ptrBatch = reinterpret_cast<T *>(vptrBatch);
        SpaceToBatchHelper<NUM_BLOCK_DIMS, B2S>::run(ptrSpace, space_shape, space_strides, block_shape, pad_start, block_offsets, ptrBatch, batch_shape, batch_strides);
    };

    Nd4jStatus _batchToSpace(nd4j::LaunchContext * context, int internal_block_dims, NDArray *input, NDArray *output, std::vector<Nd4jLong> &internal_input_shape, std::vector<Nd4jLong> &internal_output_shape, Nd4jLong *block_shape, Nd4jLong *crops) {

        return Status::OK();
    }

#define STB_DIM (0, 1),\
                (1, 2),\
                (2, 3),\
                (3, 4)

#define STB_BOOL (0, false),\
                 (1, true)

    BUILD_TRIPLE_TEMPLATE(template void _execute, (nd4j::LaunchContext * context, void *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, void *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides), LIBND4J_TYPES, STB_DIM, STB_BOOL);

#undef STB_BOOL
#undef STB_DIM
*/

}
}
}